#include "hip/hip_runtime.h"
#include "fields.h"
#include "get_error.h"

Fields::Fields(Parameters* pars, Grids* grids) :
  size_(sizeof(hipComplex)*grids->NxNycNz), N(grids->NxNycNz), pars_(pars), grids_(grids),
  phi(nullptr), phi_h(nullptr), apar(nullptr), apar_h(nullptr)
{
  checkCuda(hipMalloc((void**) &phi, size_));

  int nn = grids->NxNycNz; int nt = min(nn, 512); int nb = 1 + (nn-1)/nt;  hipComplex zero = make_hipComplex(0.,0.);
  setval <<< nb, nt >>> (phi, zero, nn);

  //  hipMemset(phi, 0., size_);

  hipHostMalloc((void**) &phi_h, size_);

  if (pars_->beta > 0.) {
    checkCuda(hipMalloc((void**) &apar, size_));

    hipMemset(apar, 0., size_); setval <<< nb, nt >>> (apar, zero, nn);

    hipHostMalloc((void**) &apar_h, size_);
  }
}

Fields::~Fields() {
  if (phi)     hipFree(phi);
  if (phi_h)   hipHostFree(phi_h);
  if (apar)    hipFree(apar);
  if (apar_h)  hipHostFree(apar_h);
}

void Fields::print_phi(void)
{
  CP_TO_CPU(phi_h, phi, size_);
  printf("\n");
  for (int j=0; j<N; j++) printf("phi(%d) = (%e, %e) \n",j, phi_h[j].x, phi_h[j].y);
  printf("\n");
}

void Fields::print_apar(void)
{
  CP_TO_CPU(apar_h, apar, size_);
  printf("\n");
  for (int j=0; j<N; j++) printf("apar(%d) = (%e, %e) \n",j, apar_h[j].x, apar_h[j].y);
  printf("\n");
}

void Fields::rescale(float * phi_max) {
  int nn1 = grids_->NxNyc; int nt1 = min(nn1, 32); int nb1 = 1 + (nn1-1)/nt1;
  int nn2 = grids_->Nz;    int nt2 = min(nn2, 32); int nb2 = 1 + (nn2-1)/nt2;
  dim3 dB, dG;
  dB = dim3(nt1, nt2, 1);
  dG = dim3(nb1, nb2, 1);
  rescale_kernel <<< dG, dB >>> (phi, phi_max, 1);
}

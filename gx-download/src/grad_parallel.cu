#include "hip/hip_runtime.h"
#include "grad_parallel.h"
#define GGPF <<< dGf, dBf >>> 
#define GGP2 <<< dGd, dBd >>> 
#define GGP <<< dG, dB >>>

GradParallelPeriodic::GradParallelPeriodic(Grids* grids) :
  grids_(grids)
{
  // (ky, kx, theta) <-> (ky, kx, kpar)
  hipfftCreate(&zft_plan_forward);
  hipfftCreate(&zft_plan_inverse);
  hipfftCreate(&dz_plan_forward);
  hipfftCreate(&dz_plan_inverse);
  hipfftCreate(&abs_dz_plan_forward);

  int n = grids_->Nz; 			// size of FFT
  int isize = grids_->NxNycNz;		// size of input data
  int osize = grids_->NxNycNz;		// size of output data
  int dim = 1;				// 1 dimensional
  int istride = grids_->NxNyc;		// distance between two elements in a batch 
					// = distance between (ky,kx,z=1) and (ky,kx,z=2) = Nx*(Ny/2+1)
  int idist = 1;			// idist = distance between first element of consecutive batches 
					// = distance between (ky=1,kx=1,z=1) and (ky=2,kx=1,z=1) = 1
  int ostride = grids_->NxNyc;
  int odist = 1;
  int batchsize = grids_->NxNyc;	// number of consecutive transforms
  size_t workSize;

  hipfftMakePlanMany(zft_plan_forward, dim, &n, &isize, istride, idist, &osize, ostride, odist, HIPFFT_C2C, batchsize, &workSize);
  hipfftMakePlanMany(zft_plan_inverse, dim, &n, &isize, istride, idist, &osize, ostride, odist, HIPFFT_C2C, batchsize, &workSize);
  hipfftMakePlanMany( dz_plan_forward, dim, &n, &isize, istride, idist, &osize, ostride, odist, HIPFFT_C2C, batchsize, &workSize);
  hipfftMakePlanMany( dz_plan_inverse, dim, &n, &isize, istride, idist, &osize, ostride, odist, HIPFFT_C2C, batchsize, &workSize);
  hipfftMakePlanMany(abs_dz_plan_forward,
		                     dim, &n, &isize, istride, idist, &osize, ostride, odist, HIPFFT_C2C, batchsize, &workSize);

  // set up callback functions
  hipDeviceSynchronize();
  hipfftXtSetCallback(   zft_plan_forward, (void**)   &zfts_callbackPtr, HIPFFT_CB_ST_COMPLEX, (void**)&grids_->kz);
  hipfftXtSetCallback(    dz_plan_forward, (void**)   &i_kz_callbackPtr, HIPFFT_CB_ST_COMPLEX, (void**)&grids_->kzp);
  hipfftXtSetCallback(abs_dz_plan_forward, (void**) &abs_kz_callbackPtr, HIPFFT_CB_ST_COMPLEX, (void**)&grids_->kzp);
  hipDeviceSynchronize();

  int nn1, nt1, nb1, nn2, nt2, nb2, nn3, nt3, nb3;

  nn1 = grids_->NxNyc;                          nt1 = min(nn1, 32);         nb1 = 1 + (nn1-1)/nt1;
  nn2 = grids_->Nz;                             nt2 = min(nn2, 32);         nb2 = 1 + (nn2-1)/nt2;
  nn3 = grids_->Nmoms*grids_->Nspecies;         nt3 = min(nn3, 1);          nb3 = 1 + (nn3-1)/nt3;

  dBd = dim3(nt1, nt2, nt3);
  dGd = dim3(nb1, nb2, nb3);
  
  nn1 = grids_->NxNyc;                          nt1 = min(nn1, 32);         nb1 = 1 + (nn1-1)/nt1;
  nn2 = grids_->Nz;                             nt2 = min(nn2, 32);         nb2 = 1 + (nn2-1)/nt2;

  dBf = dim3(nt1, nt2, 1);
  dGf = dim3(nb1, nb2, 1);
  
}

GradParallelPeriodic::~GradParallelPeriodic() {
  hipfftDestroy(zft_plan_forward);
  hipfftDestroy(zft_plan_inverse);
  hipfftDestroy( dz_plan_forward);
  hipfftDestroy( dz_plan_inverse);
  hipfftDestroy(abs_dz_plan_forward);
}

// Dealias in kz
void GradParallelPeriodic::dealias(MomentsG* G)
{
  for (int i = 0; i < grids_->Nmoms*grids_->Nspecies; i++) hipfftExecC2C(zft_plan_forward, G->G(i), G->G(i), HIPFFT_FORWARD);
  kz_dealias GGP2 (G->G(), grids_->kzm, grids_->Nmoms*grids_->Nspecies);
  for (int i = 0; i < grids_->Nmoms*grids_->Nspecies; i++) hipfftExecC2C(zft_plan_inverse, G->G(i), G->G(i), HIPFFT_BACKWARD);  
}

// Dealias in kz
void GradParallelPeriodic::dealias(hipComplex* f)
{
  hipfftExecC2C(zft_plan_forward, f, f, HIPFFT_FORWARD);
  int one  = 1;
  kz_dealias GGPF (f, grids_->kzm, one);
  hipfftExecC2C(zft_plan_inverse, f, f, HIPFFT_BACKWARD);  
}

// Fourier transform all moments 
void GradParallelPeriodic::zft(MomentsG* G)
{
  // for now, loop over all l and m because cannot batch 
  for(int i = 0; i < grids_->Nmoms*grids_->Nspecies; i++) hipfftExecC2C(zft_plan_forward, G->G(i), G->G(i), HIPFFT_FORWARD);
}

void GradParallelPeriodic::zft_inverse(MomentsG* G)
{
  // for now, loop over all l and m because cannot batch 
  for(int i = 0; i < grids_->Nmoms*grids_->Nspecies; i++) hipfftExecC2C(zft_plan_inverse, G->G(i), G->G(i), HIPFFT_BACKWARD);
}

// Fourier transform for a single moment
void GradParallelPeriodic::zft(hipComplex* mom, hipComplex* res)
{
  hipfftExecC2C(zft_plan_forward, mom, res, HIPFFT_FORWARD);
}
/*
// inverse Fourier transform for a single moment
void GradParallelPeriodic::zft_inverse(hipComplex* mom, hipComplex* res)
{
  hipfftExecC2C(zft_plan_inverse, mom, res, HIPFFT_BACKWARD);
}
*/
// FFT and derivative for all moments
void GradParallelPeriodic::dz(MomentsG* G)
{
  // FFT and derivative on parallel term
  // i*kz*G calculated via callback, defined as part of dz_plan_forward
  // for now, loop over all l and m because cannot batch 
  // eventually will optimize by first transposing so that z is fastest index

  for(int i = 0; i < grids_->Nmoms*grids_->Nspecies; i++) {
    // forward FFT (z -> kz) & multiply by i kz (via callback)
    hipfftExecC2C(dz_plan_forward, G->G(i), G->G(i), HIPFFT_FORWARD);

    // backward FFT (kz -> z)
    hipfftExecC2C(dz_plan_inverse, G->G(i), G->G(i), HIPFFT_BACKWARD);
  }
}

// FFT and derivative for a single moment
void GradParallelPeriodic::dz(hipComplex* mom, hipComplex* res)
{
  hipfftExecC2C(dz_plan_forward, mom, res, HIPFFT_FORWARD);
  hipfftExecC2C(dz_plan_inverse, res, res, HIPFFT_BACKWARD);
}

// FFT and |kz| operator for a single moment
void GradParallelPeriodic::abs_dz(hipComplex* mom, hipComplex* res)
{
  hipfftExecC2C(abs_dz_plan_forward, mom, res, HIPFFT_FORWARD);
  hipfftExecC2C(dz_plan_inverse, res, res, HIPFFT_BACKWARD);
}

// FFT only for a single moment -- deprecated. Should change to zft, dropping dir parameter
void GradParallelPeriodic::fft_only(hipComplex* mom, hipComplex* res, int dir)
{
  // use dz_plan_inverse since it does not multiply by i kz via callback 
  hipfftExecC2C(dz_plan_inverse, mom, res, dir);
}

GradParallelLocal::GradParallelLocal(Grids* grids) :
  grids_(grids)
{
  dB = 512;
  dG = 1 + (grids_->NxNycNz-1)/dB.x;
}

void GradParallelLocal::dz(MomentsG *G)
{
  G->scale(make_hipComplex(0.,1.));
}

void GradParallelLocal::zft(MomentsG *G) {return;}
void GradParallelLocal::zft(hipComplex* mom, hipComplex* res) {
  scale_singlemom_kernel GGP (res, mom, make_hipComplex(1.,0.));
}
void GradParallelLocal::zft_inverse(MomentsG *G) {return;}
//void GradParallelLocal::zft_inverse(MomentsG *G, hipComplex* res) {return;}

// single moment
void GradParallelLocal::dz(hipComplex* mom, hipComplex* res) {
  scale_singlemom_kernel GGP (res, mom, make_hipComplex(0.,1.));
}
// single moment
void GradParallelLocal::abs_dz(hipComplex* mom, hipComplex* res) {
  scale_singlemom_kernel GGP (res, mom, make_hipComplex(1.,0.));
}


GradParallel1D::GradParallel1D(Grids* grids) :
  grids_(grids)
{
  // (theta) <-> (kpar)
  hipfftCreate(&dz_plan_forward);
  hipfftCreate(&dz_plan_inverse);

  // MFM: Plan for 1d FFT
  hipfftPlan1d(&dz_plan_forward, grids_->Nz, HIPFFT_R2C, 1);
  hipfftPlan1d(&dz_plan_inverse, grids_->Nz, HIPFFT_C2R, 1);

  hipDeviceSynchronize();
  hipfftXtSetCallback(dz_plan_forward, (void**) &i_kz_1d_callbackPtr, HIPFFT_CB_ST_COMPLEX, (void**)&grids_->kz);
  hipDeviceSynchronize();

  hipMalloc((void**) &b_complex, sizeof(hipComplex)*(grids_->Nz/2+1));
}

GradParallel1D::~GradParallel1D() {
  hipfftDestroy(dz_plan_forward);
  hipfftDestroy(dz_plan_inverse);
  hipFree(b_complex);
}

void GradParallel1D::dz1D(float* b)  // even tho cuda 11+ overwrites inputs, this is ok
{
  hipfftExecR2C(dz_plan_forward, b, b_complex); 
  hipfftExecC2R(dz_plan_inverse, b_complex, b);
}


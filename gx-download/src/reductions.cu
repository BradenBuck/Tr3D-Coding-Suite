#include "hip/hip_runtime.h"
#include "reductions.h"
#include <iostream>

// ======= Grid_Species_Reduce ==========
Grid_Species_Reduce::Grid_Species_Reduce(Grids *grids, std::vector<int> spectra) :
  grids_(grids), spectra_(spectra)
{
  Addwork = nullptr;     sizeWork = 0;         sizeAdd = 0;

  int J;  J = spectra_.size();
  initialized.assign(J, 0);   desc.resize(J);   extents.resize(J);
  
  extent['y'] = grids_->Nyc;
  extent['x'] = grids_->Nx;
  extent['z'] = grids_->Nz;
  extent['s'] = grids_->Nspecies;;

  for (auto mode : Pmode) extent_P.push_back(extent[mode]);;
  for (int j = 0; j < J; j++) {
    if (spectra_[j] == 1) {
      for (auto mode : pModes[j]) extents[j].push_back(extent[mode]);

      cutensorInit(&handle);
      hiptensorInitTensorDescriptor(&handle, &dP, nPmode, extent_P.data(), NULL, cfloat, HIPTENSOR_OP_ABS);
      hiptensorInitTensorDescriptor(&handle, &desc[j], pModes[j].size(), extents[j].data(), NULL, cfloat, HIPTENSOR_OP_ABS);
    }
  }
}

Grid_Species_Reduce::~Grid_Species_Reduce()
{
  if (Addwork) hipFree(Addwork);
}

// ======== Grid_Species_Reduce  ==============
void Grid_Species_Reduce::Sum(float* P2, float* res, int ispec)
{
  if (initialized[ispec] == 0) {
  
    cutensorReductionGetWorkspace(&handle, P2, &dP, Pmode.data(),
				  res, &desc[ispec], pModes[ispec].data(),
				  res, &desc[ispec], pModes[ispec].data(),
				  opAdd, typeCompute, &sizeAdd);
    if (sizeAdd > sizeWork) {
      sizeWork = sizeAdd;
      if (Addwork) hipFree (Addwork);
      if (hipSuccess != hipMalloc(&Addwork, sizeWork)) {
	Addwork = nullptr;	sizeWork = 0;
      }
    }
    initialized[ispec]  = 1;
  }
  
  hiptensorReduction(&handle,
		    (const void*) &alpha, P2, &dP, Pmode.data(),
		    (const void*) &beta,  res,  &desc[ispec], pModes[ispec].data(),
		    res,  &desc[ispec], pModes[ispec].data(),
		    opAdd, typeCompute, Addwork, sizeWork, 0);
}		     

// ======= Grid_Reduce ==========
Grid_Reduce::Grid_Reduce(Grids *grids, std::vector<int> spectra) :
  grids_(grids), spectra_(spectra)
{
  Addwork = nullptr;     sizeWork = 0;         sizeAdd = 0;

  int J;  J = spectra_.size();
  initialized.assign(J, 0);   desc.resize(J);   extents.resize(J);
  sizeWork = 0;               sizeAdd = 0;          

  extent['y'] = grids_->Nyc;
  extent['x'] = grids_->Nx;
  extent['z'] = grids_->Nz;
  
  for (auto mode : Imode) extent_I.push_back(extent[mode]);
  for (int j = 0; j < J; j++) {
    if (spectra_[j] == 1) {
      for (auto mode : iModes[j]) extents[j].push_back(extent[mode]);

      //      printf("0 =? %d \n",iModes[0].size());
      
      cutensorInit(&handle);
      hiptensorInitTensorDescriptor(&handle, &dI, nImode, extent_I.data(), NULL, cfloat, HIPTENSOR_OP_ABS);
      hiptensorInitTensorDescriptor(&handle, &desc[j], iModes[j].size(), extents[j].data(), NULL, cfloat, HIPTENSOR_OP_ABS);

    }
  }
}

Grid_Reduce::~Grid_Reduce()
{
  if (Addwork) hipFree(Addwork);
}

void Grid_Reduce::Sum(float* I2, float* res, int ispec)
{

  if (initialized[ispec] == 0) {
  
    cutensorReductionGetWorkspace(&handle, I2, &dI, Imode.data(),
				  res, &desc[ispec], iModes[ispec].data(),
				  res, &desc[ispec], iModes[ispec].data(),
				  opAdd, typeCompute, &sizeAdd);
    if (sizeAdd > sizeWork) {
      sizeWork = sizeAdd;
      if (Addwork) hipFree (Addwork);
      if (hipSuccess != hipMalloc(&Addwork, sizeWork)) {
	Addwork = nullptr;	sizeWork = 0;
      }
    }
    initialized[ispec]  = 1;
  } 

  hiptensorReduction(&handle,
		    (const void*) &alpha, I2, &dI, Imode.data(),
		    (const void*) &beta,  res,  &desc[ispec], iModes[ispec].data(),
		    res,  &desc[ispec], iModes[ispec].data(),
		    opAdd, typeCompute, Addwork, sizeWork, 0);
}

// ======= All_Reduce ==========
All_Reduce::All_Reduce(Grids *grids, std::vector<int> spectra) :
  grids_(grids), spectra_(spectra)
{
  Addwork = nullptr;      sizeWork = 0;         sizeAdd = 0;

  int J;  J = spectra_.size();
  initialized.assign(J, 0);   desc.resize(J);   extents.resize(J);
  
  extent['y'] = grids_->Nyc;
  extent['x'] = grids_->Nx;
  extent['z'] = grids_->Nz;
  extent['l'] = grids_->Nl;
  extent['m'] = grids_->Nm;
  extent['s'] = grids_->Nspecies;;

  for (auto mode : Wmode) extent_W.push_back(extent[mode]);;

  // Build tensor descriptions for partial summations here
  for (int j = 0; j < J; j++) {
    if (spectra_[j] == 1) {
      for (auto mode : Modes[j]) extents[j].push_back(extent[mode]);

      cutensorInit(&handle);
      hiptensorInitTensorDescriptor(&handle, &dW, nWmode, extent_W.data(), NULL, cfloat, HIPTENSOR_OP_ABS);
      hiptensorInitTensorDescriptor(&handle, &desc[j], Modes[j].size(), extents[j].data(), NULL, cfloat, HIPTENSOR_OP_ABS);
    }
  }
}

All_Reduce::~All_Reduce()
{
  if (Addwork) hipFree(Addwork);
}

void All_Reduce::Sum(float* W, float* res, int ispec) 
{

  if (initialized[ispec] == 0) {

    // Get size of workspace that will be used, stored in sizeAdd (sizeWork)
    cutensorReductionGetWorkspace(&handle, W, &dW, Wmode.data(),
				  res, &desc[ispec], Modes[ispec].data(),
				  res, &desc[ispec], Modes[ispec].data(),
				  opAdd, typeCompute, &sizeAdd);
  
    // if the size is larger than currently allocated (starting with unallocated) space, free
    // the old one (if it is allocated) and allocate the larger space
    // Assume it is fine to use the larger work space freely. 
    //    printf("Workspace allocation: %d \t with size %d \n",ispec,sizeAdd);
    if (sizeAdd > sizeWork) {
      sizeWork = sizeAdd;
      if (Addwork) hipFree (Addwork);
      if (hipSuccess != hipMalloc(&Addwork, sizeWork)) {
	Addwork = nullptr;	sizeWork = 0;
      }
      //      printf("work size = %d \n", sizeWork);
    }
    initialized[ispec] = 1;
  } 

  //  printf("Reduction: %d \n",ispec);
  hiptensorReduction(&handle,
		    (const void*) &alpha, W,   &dW,          Wmode.data(),
		    (const void*) &beta,  res, &desc[ispec], Modes[ispec].data(),
		    res,  &desc[ispec], Modes[ispec].data(),
		    opAdd, typeCompute, Addwork, sizeWork, 0);
  // The final argument in this call is the stream used for the calculation
}

//============ Block_Reduce ==============
Block_Reduce::Block_Reduce(int N) : N_(N)
{
  Addwork = nullptr;      sizeAdd = 0;
  Maxwork = nullptr;      sizeMax = 0;

  extent['a'] = N_;
  extent['s'] = 1;
  for (auto mode : Amode) extent_A.push_back(extent[mode]); // incoming tensor assuming data is contiguous
  for (auto mode : Bmode) extent_B.push_back(extent[mode]); // target scalar output

  cutensorInit(&handle);
  hiptensorInitTensorDescriptor(&handle, &dA, nAmode, extent_A.data(), NULL, cfloat, HIPTENSOR_OP_ABS);
  hiptensorInitTensorDescriptor(&handle, &dB, nBmode, extent_B.data(), NULL, cfloat, HIPTENSOR_OP_ABS);
}

Block_Reduce::~Block_Reduce()
{
  if (Addwork) hipFree(Addwork);
  if (Maxwork) hipFree(Maxwork);  
}

void Block_Reduce::Max(float* A2, float* B)
{
  // calculate reduction, B = max(|A2|), over first few indices

  if (first_Max) {
    
    // get workspace (sizeMax) for a max (opMax) over |P2|
    cutensorReductionGetWorkspace(&handle,
				  A2, &dA, Amode.data(),
				  B,  &dB, Bmode.data(),
				  B,  &dB, Bmode.data(),
				  opMax, typeCompute, &sizeMax);
    
    if (hipSuccess != hipMalloc(&Maxwork, sizeMax)) {
      Maxwork = nullptr;      sizeMax = 0;
    }
    first_Max = false;
  }

  hiptensorReduction(&handle,
		    (const void*) &alpha, A2, &dA, Amode.data(),
		    (const void*) &beta,  B,  &dB, Bmode.data(),
		    B,  &dB, Bmode.data(),
		    opMax, typeCompute, Maxwork, sizeMax, 0);
}

void Block_Reduce::Sum(float* A, float* B, int i)
{
  // calculate full reduction, B = sum A

  if (first_Sum) {
    
    cutensorReductionGetWorkspace(&handle,
				  A,  &dA, Amode.data(),
				  B,  &dB, Bmode.data(),
				  B,  &dB, Bmode.data(),
				  opAdd, typeCompute, &sizeAdd);
    
    if (hipSuccess != hipMalloc(&Addwork, sizeAdd)) {
      Addwork = nullptr;      sizeAdd = 0;
    }
    first_Sum = false;
  }

  hiptensorReduction(&handle,
		    (const void*) &alpha, A, &dA, Amode.data(),
		    (const void*) &beta,  B, &dB, Bmode.data(),
		    B, &dB, Bmode.data(),
		    opAdd, typeCompute, Addwork, sizeAdd, 0);
}

//============ dBlock_Reduce (double precision) ==============
dBlock_Reduce::dBlock_Reduce(int N) : N_(N)
{
  Addwork = nullptr;      sizeAdd = 0;
  Maxwork = nullptr;      sizeMax = 0;

  extent['a'] = N_;
  extent['s'] = 1;
  for (auto mode : Amode) extent_A.push_back(extent[mode]); // incoming tensor assuming data is contiguous
  for (auto mode : Bmode) extent_B.push_back(extent[mode]); // target scalar output

  cutensorInit(&handle);
  hiptensorInitTensorDescriptor(&handle, &dA, nAmode, extent_A.data(), NULL, dfloat, HIPTENSOR_OP_ABS);
  hiptensorInitTensorDescriptor(&handle, &dB, nBmode, extent_B.data(), NULL, dfloat, HIPTENSOR_OP_ABS);
}

dBlock_Reduce::~dBlock_Reduce()
{
  if (Addwork) hipFree(Addwork);
  if (Maxwork) hipFree(Maxwork);  
}

void dBlock_Reduce::Max(double* A2, double* B)
{
  // calculate reduction, B = max(|A2|), over first few indices

  if (first_Max) {
    
    // get workspace (sizeMax) for a max (opMax) over |P2|
    cutensorReductionGetWorkspace(&handle,
				  A2, &dA, Amode.data(),
				  B,  &dB, Bmode.data(),
				  B,  &dB, Bmode.data(),
				  opMax, typeCompute64, &sizeMax);
    
    if (hipSuccess != hipMalloc(&Maxwork, sizeMax)) {
      Maxwork = nullptr;      sizeMax = 0;
    }
    first_Max = false;
  }

  hiptensorReduction(&handle,
		    (const void*) &alpha64, A2, &dA, Amode.data(),
		    (const void*) &beta64,  B,  &dB, Bmode.data(),
		    B,  &dB, Bmode.data(),
		    opMax, typeCompute64, Maxwork, sizeMax, 0);
}

void dBlock_Reduce::Sum(double* A, double* B, int i)
{
  // calculate full reduction, B = sum A

  if (first_Sum) {
    
    cutensorReductionGetWorkspace(&handle,
				  A,  &dA, Amode.data(),
				  B,  &dB, Bmode.data(),
				  B,  &dB, Bmode.data(),
				  opAdd, typeCompute64, &sizeAdd);
    
    if (hipSuccess != hipMalloc(&Addwork, sizeAdd)) {
      Addwork = nullptr;      sizeAdd = 0;
    }
    first_Sum = false;
  }

  hiptensorReduction(&handle,
		    (const void*) &alpha64, A, &dA, Amode.data(),
		    (const void*) &beta64,  B, &dB, Bmode.data(),
		    B, &dB, Bmode.data(),
		    opAdd, typeCompute64, Addwork, sizeAdd, 0);
  /*
  double * vec;
  hipHostMalloc((void **) &vec, sizeof(double)*N_);
  CP_TO_CPU(vec, A, sizeof(double)*N_);
  for (int n=0; n<N_; n++) {
    printf("A[%d] = %e \n",n,vec[n]);
  }

  CP_TO_CPU(vec, B, sizeof(double));
  printf("B = %e \n",vec[0]);
  hipHostFree(vec);
  */
}

//============ Species_Reduce ==============
Species_Reduce::Species_Reduce(int N, int nspecies) : N_(N)
{
  Addwork = nullptr;     sizeAdd = 0; 
  
  extent['a'] = N_;
  extent['s'] = nspecies;
  for (auto mode : Qmode) extent_Q.push_back(extent[mode]); // incoming tensor without abs value
  for (auto mode : Rmode) extent_R.push_back(extent[mode]); // target species scalar output

  cutensorInit(&handle);  
  hiptensorInitTensorDescriptor(&handle, &dQ, nQmode, extent_Q.data(), NULL, cfloat, HIPTENSOR_OP_IDENTITY);
  hiptensorInitTensorDescriptor(&handle, &dR, nRmode, extent_R.data(), NULL, cfloat, HIPTENSOR_OP_IDENTITY);
}

Species_Reduce::~Species_Reduce()
{
  if (Addwork) hipFree(Addwork);
}

void Species_Reduce::Sum(float* Q, float* R, int i)
{
  // calculate reduction, R = sum Q, leaving results sorted by species only

  if (first_Sum) {
    
    // get workspace (sizeAdd) for a sum (opSum) over Q
    cutensorReductionGetWorkspace(&handle,
				  Q,  &dQ, Qmode.data(),
				  R,  &dR, Rmode.data(),
				  R,  &dR, Rmode.data(),
				  opAdd, typeCompute, &sizeAdd);
    
    if (hipSuccess != hipMalloc(&Addwork, sizeAdd)) {
      Addwork = nullptr;      sizeAdd = 0;
    }
    first_Sum = false;
  }

  hiptensorReduction(&handle,
		    (const void*) &alpha, Q, &dQ, Qmode.data(),
		    (const void*) &beta,  R, &dR, Rmode.data(),
		    R, &dR, Rmode.data(),
		    opAdd, typeCompute, Addwork, sizeAdd, 0);
}

//============ DenseM ==============
DenseM::DenseM(int N, int M) : N_(N), M_(M)
{ 
  Multwork = nullptr;      sizeWork = 0;

  extent['g'] = M_;
  extent['r'] = N_;
  extent['s'] = N_;
  for (auto mode : Mmode) extent_M.push_back(extent[mode]); 
  for (auto mode : Vmode) extent_V.push_back(extent[mode]); 
  for (auto mode : Ymode) extent_Y.push_back(extent[mode]); 
  for (auto mode : Zmode) extent_Z.push_back(extent[mode]); 
  for (auto mode : Wmode) extent_W.push_back(extent[mode]); 
  for (auto mode : Xmode) extent_X.push_back(extent[mode]); 

  cutensorInit(&handle);
  hiptensorInitTensorDescriptor(&handle, &dY, nYmode, extent_Y.data(), NULL, dfloat, HIPTENSOR_OP_IDENTITY);
  hiptensorInitTensorDescriptor(&handle, &dM, nMmode, extent_M.data(), NULL, dfloat, HIPTENSOR_OP_IDENTITY);
  hiptensorInitTensorDescriptor(&handle, &dV, nVmode, extent_V.data(), NULL, dfloat, HIPTENSOR_OP_IDENTITY);
  hiptensorInitTensorDescriptor(&handle, &dX, nXmode, extent_X.data(), NULL, dfloat, HIPTENSOR_OP_IDENTITY);
  hiptensorInitTensorDescriptor(&handle, &dW, nWmode, extent_W.data(), NULL, dfloat, HIPTENSOR_OP_IDENTITY);
  hiptensorInitTensorDescriptor(&handle, &dZ, nZmode, extent_Z.data(), NULL, dfloat, HIPTENSOR_OP_IDENTITY);

  cutensorInitContractionFind(&handle, &find, HIPTENSOR_ALGO_DEFAULT);
}

DenseM::~DenseM()
{
  if (Multwork) hipFree(Multwork);  
}

void DenseM::MatMat(double* Res, double* M1, double* M2)
{
  if (first_MM) {
    
    uint32_t alignM1, alignM2, alignRes;
    cutensorGetAlignmentRequirement(&handle, M1,  &dW, &alignM1);
    cutensorGetAlignmentRequirement(&handle, M2,  &dZ, &alignM2);
    cutensorGetAlignmentRequirement(&handle, Res, &dX, &alignRes);

    cutensorInitContractionDescriptor (&handle, &dMM, 
				       &dW, Wmode.data(), alignM1,
				       &dZ, Zmode.data(), alignM2,
				       &dX, Xmode.data(), alignRes,
				       &dX, Xmode.data(), alignRes,
				       typeCompute64);

    cutensorContractionGetWorkspace(&handle, &dMM, &find, CUTENSOR_WORKSPACE_RECOMMENDED, &sizeMM );
    if (sizeMM > 0) {
      if (hipSuccess != hipMalloc(&MMwork, sizeMM)) {MMwork = nullptr; sizeMM = 0;}
    }
    first_MM = false;

    cutensorInitContractionPlan(&handle, &MMplan, &dMM, &find, sizeMM);
  }
  
  hiptensorContraction(&handle,
		      &MMplan, (void*) &alpha64, M1, M2, (void*) &beta64, Res, Res, MMwork, sizeMM, 0);
  
}

// Res[M] = Mat[M x N] * Vec[N]
// and in terms of these tensor descriptors Y = M V
void DenseM::MatVec(double* Res, double* Mat, double* Vec)
{
  if (first_MV) {
    
    uint32_t alignVec, alignMat, alignRes;
    cutensorGetAlignmentRequirement(&handle, Mat, &dM, &alignMat);
    cutensorGetAlignmentRequirement(&handle, Vec, &dV, &alignVec);
    cutensorGetAlignmentRequirement(&handle, Res, &dY, &alignRes);

    cutensorInitContractionDescriptor (&handle, &dMV, 
				       &dM, Mmode.data(), alignMat,
				       &dV, Vmode.data(), alignVec,
				       &dY, Ymode.data(), alignRes,
				       &dY, Ymode.data(), alignRes,
				       typeCompute64);

    cutensorContractionGetWorkspace(&handle, &dMV, &find, CUTENSOR_WORKSPACE_RECOMMENDED, &sizeWork);
    if (sizeWork > 0) {
      if (hipSuccess != hipMalloc(&Multwork, sizeWork)) {Multwork = nullptr; sizeWork = 0;}
    }
    first_MV = false;

    cutensorInitContractionPlan(&handle, &MVplan, &dMV, &find, sizeWork);
  }
  
  hiptensorContraction(&handle, &MVplan, (void*) &alpha64, Mat, Vec,
		      (void*) &beta64, Res, Res, Multwork, sizeWork, 0);
  
}


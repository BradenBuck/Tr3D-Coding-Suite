#include "hip/hip_runtime.h"
#include "geometry.h"
#define GGEO <<< dimGrid, dimBlock >>>

#include <fstream>
#include <iostream>
#include <string>
#include <sstream>

Geometry::Geometry() {

  operator_arrays_allocated_=false;

  z_h          = nullptr;  gbdrift_h  = nullptr;  grho_h     = nullptr;  cvdrift_h  = nullptr;
  bmag_h       = nullptr;  bmagInv_h  = nullptr;  bgrad_h    = nullptr;  gds2_h     = nullptr;
  gds21_h      = nullptr;  gds22_h    = nullptr;  cvdrift0_h = nullptr;  gbdrift0_h = nullptr;
  jacobian_h   = nullptr;

  z            = nullptr;  gbdrift    = nullptr;  grho       = nullptr;  cvdrift    = nullptr;
  bmag         = nullptr;  bmagInv    = nullptr;  bgrad      = nullptr;  gds2       = nullptr;
  gds21        = nullptr;  gds22      = nullptr;  cvdrift0   = nullptr;  gbdrift0   = nullptr;
  jacobian     = nullptr;

  gradpar_arr  = nullptr;  Rplot      = nullptr;  Zplot      = nullptr;  aplot      = nullptr;
  Xplot        = nullptr;  Yplot      = nullptr;  Rprime     = nullptr;  Zprime     = nullptr;
  aprime       = nullptr;  deltaFL    = nullptr; 
  
  bmag_complex = nullptr;  bgrad_temp = nullptr; 
    
  // operator arrays
  kperp2       = nullptr;  omegad     = nullptr;  cv_d       = nullptr;   gb_d      = nullptr;
  kperp2_h     = nullptr; 

}

Geometry::~Geometry() {
  if (z)         hipFree(z);
  if (bmag)      hipFree(bmag);
  if (bmagInv)   hipFree(bmagInv);
  if (bgrad)     hipFree(bgrad);
  if (gds2);     hipFree(gds2);	
  if (gds21);    hipFree(gds21);	
  if (gds22);    hipFree(gds22);	
  if (gbdrift);  hipFree(gbdrift);	
  if (gbdrift0); hipFree(gbdrift0);	
  if (cvdrift);  hipFree(cvdrift);	
  if (cvdrift0); hipFree(cvdrift0);	
  if (grho);     hipFree(grho);	
  if (jacobian); hipFree(jacobian);	

  if (z_h)         hipHostFree(z_h);
  if (bmag_h)      hipHostFree(bmag_h);
  if (bmagInv_h)   hipHostFree(bmagInv_h);
  if (bgrad_h)     hipHostFree(bgrad_h);
  if (gds2_h);     hipHostFree(gds2_h);	
  if (gds21_h);    hipHostFree(gds21_h);	
  if (gds22_h);    hipHostFree(gds22_h);	
  if (gbdrift_h);  hipHostFree(gbdrift_h);	
  if (gbdrift0_h); hipHostFree(gbdrift0_h);	
  if (cvdrift_h);  hipHostFree(cvdrift_h);	
  if (cvdrift0_h); hipHostFree(cvdrift0_h);	
  if (grho_h);     hipHostFree(grho_h);	
  if (jacobian_h); hipHostFree(jacobian_h);	

  if(operator_arrays_allocated_) {
    if (kperp2) hipFree(kperp2);
    if (omegad) hipFree(omegad);
    if (cv_d)   hipFree(cv_d);
    if (gb_d)   hipFree(gb_d);
  }
}

S_alpha_geo::S_alpha_geo(Parameters *pars, Grids *grids) 
{
  int Nz = grids->Nz;
  float theta;
  operator_arrays_allocated_=false;
  size_t size = sizeof(float)*Nz;
  hipHostMalloc ((void**) &z_h, size);
  hipHostMalloc ((void**) &bmag_h, size);
  hipHostMalloc ((void**) &bmagInv_h, size);
  hipHostMalloc ((void**) &bgrad_h, size);
  hipHostMalloc ((void**) &gds2_h, size);
  hipHostMalloc ((void**) &gds21_h, size);
  hipHostMalloc ((void**) &gds22_h, size);
  hipHostMalloc ((void**) &gbdrift_h, size);
  hipHostMalloc ((void**) &gbdrift0_h, size);
  hipHostMalloc ((void**) &cvdrift_h, size);
  hipHostMalloc ((void**) &cvdrift0_h, size);
  hipHostMalloc ((void**) &grho_h, size);
  hipHostMalloc ((void**) &jacobian_h, size);

  //  hipHostMalloc((void**) &kperp2_h, sizeof(float)*grids->NxNycNz);
  
  hipMalloc ((void**) &z, size);
  hipMalloc ((void**) &bmag, size);
  hipMalloc ((void**) &bmagInv, size);
  hipMalloc ((void**) &bgrad, size);
  hipMalloc ((void**) &gds2, size);
  hipMalloc ((void**) &gds21, size);
  hipMalloc ((void**) &gds22, size);
  hipMalloc ((void**) &gbdrift, size);
  hipMalloc ((void**) &gbdrift0, size);
  hipMalloc ((void**) &cvdrift, size);
  hipMalloc ((void**) &cvdrift0, size);
  hipMalloc ((void**) &grho, size);
  hipMalloc ((void**) &jacobian, size);
  
  float qsf = pars->qsf;
  float beta_e = pars->beta;
  float rmaj = pars->rmaj;
  specie* species = pars->species_h;
  
  gradpar = (float) abs(1./(qsf*rmaj));
  zero_shat_ = pars->zero_shat;
  shat = pars->shat;
  pars->drhodpsi = 1.; 
  pars->kxfac = 1.;
  
  if(pars->shift < 0.) {
    pars->shift = 0.;
    for(int s=0; s<pars->nspec_in; s++) { 
      pars->shift += qsf*qsf*rmaj*beta_e*
	(species[s].temp/species[pars->nspec_in-1].temp)*
	(species[s].tprim + species[s].fprim);
    }
  }
  float shift = pars->shift;
 
  DEBUGPRINT("\n\n Using s-alpha geometry: \n\n");
  for(int k=0; k<Nz; k++) {
    z_h[k] = 2.*M_PI *pars->Zp *(k-Nz/2)/Nz;
    DEBUGPRINT("theta[%d] = %f \n",k,z_h[k]);
    if(pars->local_limit) {z_h[k] = 0.;} // outboard-midplane
    theta = z_h[k];
    
    bmag_h[k] = 1. / (1. + pars->eps * cos(theta));
    bgrad_h[k] = gradpar * pars->eps * sin(theta) * bmag_h[k]; 

    gds2_h[k] = 1. + pow((shat * theta - shift * sin(theta)), 2);
    gds21_h[k] = -shat * (shat * theta - shift * sin(theta));
    gds22_h[k] = pow(shat,2);

    gbdrift_h[k] = 1. / (2.*rmaj) *
               (cos(theta) + (shat * theta - shift * sin(theta)) * sin(theta));
    cvdrift_h[k] = gbdrift_h[k];

    gbdrift0_h[k] = - shat * sin(theta) / (2.*rmaj);
    cvdrift0_h[k] = gbdrift0_h[k];

    grho_h[k] = 1;

    if(pars->const_curv) {
      cvdrift_h[k] = 1./(2.*rmaj);
      gbdrift_h[k] = 1./(2.*rmaj);
      cvdrift0_h[k] = 0.;
      gbdrift0_h[k] = 0.;
    }
    
    if(pars->slab) {
      cvdrift_h[k] = 0.;
      gbdrift_h[k] = 0.;       
      cvdrift0_h[k] = 0.;
      gbdrift0_h[k] = 0.;
      bgrad_h[k] = 0.;
      bmag_h[k] = 1.;
    }
    if(pars->local_limit) { z_h[k] = 2 * M_PI * pars->Zp * (k-Nz/2) / Nz; }

    // calculate these derived coefficients after slab overrides
    bmagInv_h[k] = 1./bmag_h[k];
    jacobian_h[k] = 1. / abs(pars->drhodpsi * gradpar * bmag_h[k]);
  }  

  CP_TO_GPU (z,        z_h,        size);
  CP_TO_GPU (gbdrift,  gbdrift_h,  size);
  CP_TO_GPU (grho,     grho_h,     size);
  CP_TO_GPU (cvdrift,  cvdrift_h,  size);
  CP_TO_GPU (bmag,     bmag_h,     size);
  CP_TO_GPU (bmagInv,  bmagInv_h,  size);
  CP_TO_GPU (bgrad,    bgrad_h,    size);
  CP_TO_GPU (gds2,     gds2_h,     size);
  CP_TO_GPU (gds21,    gds21_h,    size);
  CP_TO_GPU (gds22,    gds22_h,    size);
  CP_TO_GPU (cvdrift0, cvdrift0_h, size);
  CP_TO_GPU (gbdrift0, gbdrift0_h, size);
  CP_TO_GPU (jacobian, jacobian_h, size);

  hipDeviceSynchronize();
  
  // initialize the drift arrays and kperp2
  initializeOperatorArrays(grids);
}

Eik_geo::Eik_geo() {

}

Gs2_geo::Gs2_geo() {

}

// MFM - 07/09/17
File_geo::File_geo(Parameters *pars, Grids *grids)
{

  operator_arrays_allocated_=false;
  size_t size = sizeof(float)*grids->Nz; 
  hipHostMalloc ((void**) &z_h, size);
  hipHostMalloc ((void**) &bmag_h, size);
  hipHostMalloc ((void**) &bmagInv_h, size);
  hipHostMalloc ((void**) &gds2_h, size);
  hipHostMalloc ((void**) &gds21_h, size);
  hipHostMalloc ((void**) &gds22_h, size);
  hipHostMalloc ((void**) &gbdrift_h, size);
  hipHostMalloc ((void**) &gbdrift0_h, size);
  hipHostMalloc ((void**) &cvdrift_h, size);
  hipHostMalloc ((void**) &cvdrift0_h, size);
  hipHostMalloc ((void**) &grho_h, size);
  hipHostMalloc ((void**) &jacobian_h, size);

  hipMalloc ((void**) &z, size);
  hipMalloc ((void**) &bmag, size);
  hipMalloc ((void**) &bmagInv, size);
  hipMalloc ((void**) &gds2, size);
  hipMalloc ((void**) &gds21, size);
  hipMalloc ((void**) &gds22, size);
  hipMalloc ((void**) &gbdrift, size);
  hipMalloc ((void**) &gbdrift0, size);
  hipMalloc ((void**) &cvdrift, size);
  hipMalloc ((void**) &cvdrift0, size);
  hipMalloc ((void**) &grho, size);
  hipMalloc ((void**) &jacobian, size);
  
  FILE * geoFile = fopen(pars->geofilename.c_str(), "r");
  
  if (geoFile == NULL) {
    printf("Cannot open file %s \n", pars->geofilename.c_str());
    exit(0);
  }

  int nlines=0;
  fpos_t lineStartPos;
  int ch;

  int ntgrid;
  int oldNz, oldnperiod;
  
  //  rewind(geoFile);
  nlines=0;
  using namespace std;
  string datline;
  ifstream myfile (pars->geofilename.c_str());
  oldNz = grids->Nz;
  int newNz = oldNz;

  if (myfile.is_open())
    {
      getline (myfile, datline);  // text
      getline (myfile, datline);  
      stringstream ss(datline);      string element;       
      getline( ss, element, ' '); ntgrid         = stoi(element);    
      getline( ss, element, ' '); pars->nperiod  = stoi(element);
      getline( ss, element, ' '); newNz          = stoi(element);   
      getline( ss, element, ' '); pars->drhodpsi = stof(element);
      getline( ss, element, ' '); pars->rmaj     = stof(element);
      getline( ss, element, ' '); pars->shat     = stof(element);
      getline( ss, element, ' '); pars->kxfac    = stof(element);       
      getline( ss, element, ' '); pars->qsf      = stof(element);       

      shat       = pars->shat;
      drhodpsi   = pars->drhodpsi;
      oldnperiod = pars->nperiod;
      
      DEBUGPRINT("\n\nIN READ_GEO_INPUT:\nntgrid = %d, nperiod = %d, Nz = %d, rmaj = %f, shat = %f\n\n\n",
		 ntgrid, pars->nperiod, grids->Nz, pars->rmaj, shat);
      
      if(oldNz != newNz) {
	printf("old Nz = %d \t new Nz = %d \n",oldNz,newNz);
	printf("You must set ntheta in the namelist equal to ntheta in the geofile. Exiting...\n");
	abort();
      }
      int Nz = newNz;
      if(oldnperiod != pars->nperiod) {
	printf("You must set nperiod in the namelist equal to nperiod in the geofile. Exiting...\n");
	abort();
      }
      
      getline (myfile, datline);  // text
      for (int idz=0; idz < newNz; idz++) {
	getline (myfile, datline); stringstream ss(datline);
	getline( ss, element, ' '); gbdrift_h[idz] = stof(element); gbdrift_h[idz] *= 0.25;
        getline( ss, element, ' '); gradpar        = stof(element);
	getline( ss, element, ' '); grho_h[idz]    = stof(element);
	getline( ss, element, ' '); z_h[idz]       = stof(element);
      }
      getline(myfile, datline); // periodic points (not always periodic, but extra)
     
      DEBUGPRINT("gbdrift[0]: %.7e    gbdrift[end]: %.7e\n",4.*gbdrift_h[0],4.*gbdrift_h[Nz-1]);
      DEBUGPRINT("z[0]: %.7e    z[end]: %.7e\n",z_h[0],z_h[Nz-1]);
      
      getline (myfile, datline);  // text
      for (int idz=0; idz < newNz; idz++) {
	getline (myfile, datline); stringstream ss(datline);
	getline( ss, element, ' '); cvdrift_h[idz] = stof(element);
	cvdrift_h[idz] *= 0.25;
        getline( ss, element, ' '); gds2_h[idz]    = stof(element);
	getline( ss, element, ' '); bmag_h[idz]    = stof(element);
	bmagInv_h[idz]  = 1./bmag_h[idz];
	jacobian_h[idz] = 1./abs(drhodpsi*gradpar*bmag_h[idz]);
      }
      getline(myfile, datline); // periodic points (not always periodic, but extra)

      DEBUGPRINT("cvdrift[0]: %.7e    cvdrift[end]: %.7e\n",4.*cvdrift_h[0],4.*cvdrift_h[Nz-1]);
      DEBUGPRINT("bmag[0]: %.7e    bmag[end]: %.7e\n",bmag_h[0],bmag_h[Nz-1]);
      DEBUGPRINT("gds2[0]: %.7e    gds2[end]: %.7e\n",gds2_h[0],gds2_h[Nz-1]);

      getline(myfile, datline); // text
      for (int idz=0; idz < newNz; idz++) {
	getline (myfile, datline); stringstream ss(datline);
	getline( ss, element, ' '); gds21_h[idz] = stof(element); 
        getline( ss, element, ' '); gds22_h[idz] = stof(element);
      }
      getline(myfile, datline); // periodic points (not always periodic, but extra)

      DEBUGPRINT("gds21[0]: %.7e    gds21[end]: %.7e\n",gds21_h[0],gds21_h[Nz-1]);
      DEBUGPRINT("gds22[0]: %.7e    gds22[end]: %.7e\n",gds22_h[0],gds22_h[Nz-1]);

            getline(myfile, datline); // text
      for (int idz=0; idz < newNz; idz++) {
	getline (myfile, datline); stringstream ss(datline);
	getline( ss, element, ' '); cvdrift0_h[idz] = stof(element); cvdrift0_h[idz] *= 0.25;
        getline( ss, element, ' '); gbdrift0_h[idz] = stof(element); gbdrift0_h[idz] *= 0.25;
      }
      getline(myfile, datline); // periodic points (not always periodic, but extra)

      DEBUGPRINT("gds21[0]: %.7e    gds21[end]: %.7e\n",gds21_h[0],gds21_h[Nz-1]);
      DEBUGPRINT("gds22[0]: %.7e    gds22[end]: %.7e\n",gds22_h[0],gds22_h[Nz-1]);
      
      myfile.close();      
    }
  else cout << "Failed to open";    
  
  //copy host variables to device variables
  CP_TO_GPU (z,        z_h,        size);
  CP_TO_GPU (gbdrift,  gbdrift_h,  size);
  CP_TO_GPU (grho,     grho_h,     size);
  CP_TO_GPU (cvdrift,  cvdrift_h,  size);
  CP_TO_GPU (bmag,     bmag_h,     size);
  CP_TO_GPU (bmagInv,  bmagInv_h,  size);
  //  CP_TO_GPU (bgrad,    bgrad_h,    size);
  CP_TO_GPU (gds2,     gds2_h,     size);
  CP_TO_GPU (gds21,    gds21_h,    size);
  CP_TO_GPU (gds22,    gds22_h,    size);
  CP_TO_GPU (cvdrift0, cvdrift0_h, size);
  CP_TO_GPU (gbdrift0, gbdrift0_h, size);
  CP_TO_GPU (jacobian, jacobian_h, size);

  hipDeviceSynchronize();

  // initialize omegad and kperp2
  initializeOperatorArrays(grids);

  // calculate bgrad
  calculate_bgrad(grids);
  CUDA_DEBUG("calc bgrad: %s \n");
}

void Geometry::initializeOperatorArrays(Grids* grids) {
  // set this flag so we know to deallocate
  operator_arrays_allocated_ = true;

  hipMalloc ((void**) &kperp2, sizeof(float)*grids->NxNycNz);
  hipMalloc ((void**) &omegad, sizeof(float)*grids->NxNycNz);
  hipMalloc ((void**) &cv_d,   sizeof(float)*grids->NxNycNz);
  hipMalloc ((void**) &gb_d,   sizeof(float)*grids->NxNycNz);
  checkCuda  (hipGetLastError());

  hipMemset (kperp2, 0., sizeof(float)*grids->NxNycNz);
  hipMemset (omegad, 0., sizeof(float)*grids->NxNycNz);
  hipMemset (cv_d,   0., sizeof(float)*grids->NxNycNz);
  hipMemset (gb_d,   0., sizeof(float)*grids->NxNycNz);
  
  dim3 dimBlock (32, 4, 4);
  dim3 dimGrid  (1+(grids->Nyc-1)/dimBlock.x, 1+(grids->Nx-1)/dimBlock.y, 1+(grids->Nz-1)/dimBlock.z);
 
  init_kperp2 GGEO (kperp2, grids->kx, grids->ky, gds2, gds21, gds22, bmagInv, shat);
  init_omegad GGEO (omegad, cv_d, gb_d, grids->kx, grids->ky, cvdrift, gbdrift, cvdrift0, gbdrift0, shat);
  /*
  CP_TO_GPU (kperp2_h,    kperp2, sizeof(float)*grids->NxNycNz);

  for (int iz=0; iz < grids->Nz; iz++) {
    for (int ikx=0; ikx < grids->Nx; ikx++) {
      for (int iky=0; iky< grids->Nyc; iky++) {
	printf("kperp2(%d,%d,%d) = %e \n", iky, ikx, iz, kperp2_h[iky + grids->Nyc*ikx + grids->Nyc*grids->Nx*iz]);
      }
      printf("\n");
    }
    printf("\n");
  }
  */  
}

// MFM - 07/25/17
void Geometry::calculate_bgrad(Grids* grids)
{
  operator_arrays_allocated_=false;

  size_t size = sizeof(float)*grids->Nz;
  hipHostMalloc((void**) &bgrad_h, size);

  hipMalloc ((void**) &bgrad, size);
  hipMalloc ((void**) &bgrad_temp, size);

  CP_ON_GPU (bgrad_temp, bmag, size);
  
  GradParallel1D* grad_par = new GradParallel1D(grids);

  //bgrad = d/dz ln(B(z)) = 1/B dB/dz
  grad_par->dz1D(bgrad_temp); // FFT and k-space derivative

  calc_bgrad <<< 1 + (grids->Nz-1)/512, 512 >>> (bgrad, bgrad_temp, bmag, gradpar);

  CP_TO_CPU (bgrad_h, bgrad, size);
  if (bgrad_temp) hipFree(bgrad_temp);

  delete grad_par;

//  for(int i=0; i<grids->Nz; i++) {
//    printf("bgrad_h[%d]: %.4e\n",i,bgrad_h[i]);
//  }
  hipDeviceSynchronize();
}


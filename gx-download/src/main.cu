#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <mpi.h>
#include <random>
#include <vector>
#include <algorithm>
#include <iostream>
#include "gx_lib.h"
#include "version.h"
#include "hip/hip_runtime_api.h"
// #include "reservoir.h"
#include "reductions.h"

int main(int argc, char* argv[])
{

  MPI_Init(&argc, &argv);
  MPI_Comm mpcom = MPI_COMM_WORLD;
  int iproc;
  MPI_Comm_rank(mpcom, &iproc);
  
  int devid = 0; // This should be determined (optionally) on the command line
  checkCuda(hipSetDevice(devid));
  hipDeviceSynchronize();

  /*  
  int N = 1000;
  int K = 3;
  
  double radius = 0.6;
  double *R;
  double *y;
  double *x;
  double *A_in;
  int *A_col;
  
  checkCuda(hipMalloc((void**) &R,  sizeof(double)*N  ) ); 
  checkCuda(hipMalloc((void**) &y,  sizeof(double)*N  ) ); 
  checkCuda(hipMalloc((void**) &x,  sizeof(double)*N*K) ); 
  
  // define the number of elements in a typical row of A to be ResDensity*N
  int nnz = K * N;
  double *A_h;
  int * A_j;
  hipHostMalloc((void**) &A_h, sizeof(double) * nnz);
  hipHostMalloc((void**) &A_j, sizeof(int)    * nnz);
  
  std::random_device rd;         std::mt19937 gen(rd()); 
  std::uniform_real_distribution<double> unif( 0., radius*2./((double) K));

  std::vector<int> col(N);     std::iota(col.begin(), col.end(), 0);
  std::vector<int> cin(K);
  
  for (int n=0; n<N; n++) {

    std::shuffle(col.begin(), col.end(), gen);
    for (int k=0; k<K; k++) cin[k] = col[k];
    std::sort(cin.begin(), cin.end());

    for (int k=0; k<K; k++) {
      A_j[k + K*n] = cin[k];
      A_h[k + K*n] = unif(gen);
      //      printf("A_j[%d] = %d \n",k+K*n, A_j[k+K*n]);
      //      printf("A_h[%d] = %e \n",k+K*n, A_h[k+K*n]);      
    }
    //    printf("\n");
  }
 
  checkCuda(hipMalloc((void**) &A_in,  sizeof(double)*nnz) ); 
  checkCuda(hipMalloc((void**) &A_col, sizeof(int)  *nnz) ); 

  CP_TO_GPU (A_in,  A_h, sizeof(double) * nnz);
  CP_TO_GPU (A_col, A_j, sizeof(int)    * nnz);

  for (int n=0; n<N; n++) {
    A_h[n] = unif(gen);
    //    printf("A_h[%d] = %e \n",n,A_h[n]);
  }    
  CP_TO_GPU (R, A_h, sizeof(double) * N);
  
  hipHostFree(A_h);
  hipHostFree(A_j);

  Red *red;
  
  red = new dBlock_Reduce(N); hipDeviceSynchronize(); 
  int nn0 = N;   int nt0 = min(nn0, 512);  int nb0 = 1 + (nn0-1)/nt0;
  int kn0 = N*K; int kt0 = min(kn0, 512);  int kb0 = 1 + (kn0-1)/kt0;
  double *x2norm;    hipMalloc((void **) &x2norm, sizeof(double)   );
  double *y2norm;    hipMalloc((void **) &y2norm, sizeof(double)   );
  double *xynorm;    hipMalloc((void **) &xynorm, sizeof(double)   );
  double *x2;        hipMalloc((void **) &x2,     sizeof(double)*N );
  double *y2;        hipMalloc((void **) &y2,     sizeof(double)*N );
  double *xy;        hipMalloc((void **) &xy,     sizeof(double)*N );

  //  setval <<< nb0, nt0 >>> (R, 1., N);
  setval <<< nb0, nt0 >>> (y, 1., N);
  setval <<< kb0, kt0 >>> (x, 1., nnz);
  
  double eval, eval_old, tol, ex, ey;
  eval=0.;  eval_old = 2.;  tol = 1.e-8;  ex = 0.;  ey = 0.;
  while (abs(eval-eval_old)/abs(eval_old) > tol) {    
    
    eval = eval_old;
    
    myPrep <<< kb0, kt0 >>> (x, R, A_col, nnz);
    mySpMV <<< nb0, nt0 >>> (x2, xy, y2, y, x, A_in, R, K, N);
    red->Sum(y2, y2norm);    red->Sum(x2, x2norm);    red->Sum(xy, xynorm);
    
    inv_scale_kernel <<< nb0, nt0 >>> (R, y, y2norm, N); 
    CP_TO_CPU(&ex, x2norm, sizeof(double));
    CP_TO_CPU(&ey, xynorm, sizeof(double));
    eval_old  = ey/ex;
    
    //    printf("eval = %e \t %e \t %e \n",eval_old,ey,ex);
    printf("eval = %e \n",eval_old);
  }
  printf(ANSI_COLOR_GREEN);
  printf("spectral radius is %e \n", eval_old);
  printf(ANSI_COLOR_RESET);
  
  // print the residual
  myPrep <<< kb0, kt0 >>> (x, R, A_col, nnz);
  mySpMV <<< nb0, nt0 >>> (x2, xy, y2, y, x, A_in, R, K, N);  
  eig_residual <<< nb0, nt0 >>> (y, A_in, x, R, x2, eval_old, K, N);
  red->Sum(x2, x2norm);  CP_TO_CPU(&ex, x2norm, sizeof(double));
  printf(ANSI_COLOR_YELLOW);  printf("RMS residual = %e \n",sqrt(ex));  printf(ANSI_COLOR_RESET);
  
  exit(1);
  */

  char *run_name;
  if ( argc < 1) {
    fprintf(stderr, "The correct usage is:\n gx <runname>\n");
    exit(1);
  } else {    
    run_name = argv[1];
    printf("Running %s \n",run_name);
  }
   
  printf("Version: %s \t Compiled: %s \n", build_git_sha, build_git_time);

  Parameters * pars         = nullptr;
  pars = new Parameters();
  pars->iproc = iproc;
  pars->get_nml_vars(run_name);
  
  Geometry    * geo         = nullptr;
  Grids       * grids       = nullptr;
  Diagnostics * diagnostics = nullptr;
  //  HermiteTransform* herm;
  
  DEBUGPRINT("Initializing grids...\n");
  grids = new Grids(pars);
  CUDA_DEBUG("Initializing grids: %s \n");

  DEBUGPRINT("Grid dimensions: Nx=%d, Ny=%d, Nz=%d, Nl=%d, Nm=%d, Nspecies=%d\n",
	     grids->Nx, grids->Ny, grids->Nz, grids->Nl, grids->Nm, grids->Nspecies);

  if(iproc==0) {
    int igeo = pars->igeo;
    DEBUGPRINT("Initializing geometry...\n");
    if(igeo==0) {
      geo = new S_alpha_geo(pars, grids);
      CUDA_DEBUG("Initializing geometry s_alpha: %s \n");
    }
    else if(igeo==1) {
      geo = new File_geo(pars, grids);
      printf("************************* \n \n \n");
      printf("Warning: assumed grho = 1 \n \n \n");
      printf("************************* \n");
      CUDA_DEBUG("Initializing geometry from file: %s \n");
    } 
    else if(igeo==2) {
      DEBUGPRINT("igeo = 2 not yet implemented!\n");
      exit(1);
      //geo = new Eik_geo();
    } 
    else if(igeo==3) {
      DEBUGPRINT("igeo = 3 not yet implemented!\n");
      exit(1);
      //geo = new Gs2_geo();
    }

    DEBUGPRINT("Initializing diagnostics...\n");
    diagnostics = new Diagnostics(pars, grids, geo);
    CUDA_DEBUG("Initializing diagnostics: %s \n");    

    //    DEBUGPRINT("Initializing Hermite transforms...\n");
    //    herm = new HermiteTransform(grids, 1); // batch size could ultimately be nspec
    //    CUDA_DEBUG("Initializing Hermite transforms: %s \n");    
  }

  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
  
  run_gx(pars, grids, geo, diagnostics);

  delete pars;
  delete grids;
  delete geo;
  delete diagnostics;

  MPI_Finalize();
  hipDeviceReset();
}

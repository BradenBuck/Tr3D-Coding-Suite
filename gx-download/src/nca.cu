#include "nca.h"

nca::nca(int N, int Nwrite) :
  N_(N), Nwrite_(Nwrite)
{
  data = nullptr;  cpu = nullptr;  tmp = nullptr; z_tmp = nullptr; tmp_d = nullptr;
  write = false;
  write_v_time = false;
  xydata = false;
  xdata = false;
  scalar = false;
  dx = false;
  d2x = false;
  adj = 1.0;
  time_start[0] = 0;
  time_start[1] = 0;
  time_start[2] = 0;
  time_start[3] = 0;
  time_start[4] = 0;
  time_start[5] = 0;

  time_count[0] = 1;
  
  if (N == 0) return;
      
  if (N > 0) {
    hipMalloc (&data, sizeof(float) * N);
    if (Nwrite > 0) {
      hipMalloc      (&tmp_d, sizeof(float) * Nwrite);  // not needed for spectra
      hipHostMalloc  (&tmp,   sizeof(float) * N);
      hipHostMalloc  (&cpu,   sizeof(float) * Nwrite);
    } else {
      hipHostMalloc  (&cpu,  sizeof(float) * N);      
    }
  } else { // omega only
    N = -N;
    if (Nwrite > 0) {
      hipHostMalloc (&z_tmp,  sizeof(hipComplex) * N);
      hipHostMalloc (&cpu,    sizeof(float)     * Nwrite);
    }
  }  
}
nca::~nca() {
  if (data)  hipFree     ( data   );
  if (tmp_d) hipFree     ( tmp_d  );
  if (tmp)   hipHostFree ( tmp    );
  if (cpu)   hipHostFree ( cpu    );
  if (z_tmp) hipHostFree ( z_tmp  );
}
void nca::increment_ts(void) {time_start[0] += 1;}

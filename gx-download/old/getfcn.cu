void getfcn(hipComplex* fcn_d)
{
  hipComplex *fcnC;
  fcnC = (hipComplex*) malloc(sizeof(hipComplex)*(Ny/2+1)*Nx*Nz);
  hipMemcpy(fcnC, fcn_d, sizeof(hipComplex)*(Ny/2+1)*Nx*Nz, hipMemcpyDeviceToHost);
  for(int k=0; k<(Nz); k++) { 
    for(int j=0; j<Nx/2+1; j++) { 
      for(int i=0; i<Ny/2+1; i++) {  
	int index = i + (Ny/2+1)*(j) + Nx*(Ny/2+1)*k;
	
	
	if(!(fcnC[index].x<.00001 && fcnC[index].y<.00001 && fcnC[index].x>-.00001 && fcnC[index].y>-.00001)) {
	
	//printf("F(%d,%d,%.2f)...
	  printf("F(%g,%g,%g)=%e + i*%e  %d\n", (float) i, (float) j, 2*M_PI*(float)(k-Nz/2)/Nz,
	                                   fcnC[index].x, fcnC[index].y, index);
        }
      }
      //printf("\n");
    }  
    for(int j=-Nx/2+1; j<0; j++) {
      for(int i=0; i<Ny/2+1; i++) {
        int index = (i) + (Ny/2+1)*(j+Nx) + Nx*(Ny/2+1)*k;
	
	
	if(!(fcnC[index].x<.00001 && fcnC[index].y<.00001 && fcnC[index].x>-.00001 && fcnC[index].y>-.00001)) {

	  printf("F(%g,%g,%g)=%e + i*%e  %d\n", (float) i, (float) j, 2*M_PI*(float)(k-Nz/2)/Nz, 
	                   fcnC[index].x, fcnC[index].y, index);
        }
      }
        
      //printf("\n");
    }
  }  
  free(fcnC);
} 

void getfcnALL(hipfftComplex* fcn_d)
{
  hipfftComplex *fcnC;
  fcnC = (hipfftComplex*) malloc(sizeof(hipfftComplex)*(Ny/2+1)*Nx*Nz);
  hipMemcpy(fcnC, fcn_d, sizeof(hipfftComplex)*(Ny/2+1)*Nx*Nz, hipMemcpyDeviceToHost);
  //for(int k=0; k<(Nz); k++) { 
    for(int j=0; j<Nx/2+1; j++) { 
      for(int i=0; i<Ny/2+1; i++) {  
	int index = i + (Ny/2+1)*(j);// + Nx*(Ny/2+1)*k;
	
	//if(!(fcnC[index].x<.00001 && fcnC[index].y<.00001 && fcnC[index].x>-.00001 && fcnC[index].y>-.00001)) {
	
	//printf("F(%d,%d,%.2f)...
	  printf("F(%g,%g)=%.5f + i*%.5f  %d  ", (float) i, (float) j, //Zp*2*M_PI*(float)(k-Nz/2)/Nz,
	                     fcnC[index].x, fcnC[index].y, index);
        //}
      }
      printf("\n");
    }  
    for(int j=-Nx/2+1; j<0; j++) {
      for(int i=0; i<Ny/2+1; i++) {
        int index = (i) + (Ny/2+1)*(j+Nx);// + Nx*(Ny/2+1)*k;
	
	//if(!(fcnC[index].x<.00001 && fcnC[index].y<.00001 && fcnC[index].x>-.00001 && fcnC[index].y>-.00001)) {

	  printf("F(%g,%g)=%.5f + i*%.5f  %d  ", (float) i, (float) j, //Zp*2*M_PI*(float)(k-Nz/2)/Nz, 
	                   fcnC[index].x, fcnC[index].y, index);
        //}
      }
        
      printf("\n");
    }
  //}  
  free(fcnC);
} 

void getfcnZCOMPLEX(hipfftComplex* fcn_d)
{
  hipfftComplex *fcnC;
  fcnC = (hipfftComplex*) malloc(sizeof(hipfftComplex)*(Ny/2+1)*Nx*Nz);
  hipMemcpy(fcnC, fcn_d, sizeof(hipfftComplex)*(Ny/2+1)*Nx*Nz, hipMemcpyDeviceToHost);
  for(int k=0; k<Nz/2+1; k++) { 
    for(int j=0; j<Nx/2+1; j++) { 
      for(int i=0; i<Ny/2+1; i++) {  
	int index = i + (Ny/2+1)*(j) + Nx*(Ny/2+1)*k;
	
	if(!(fcnC[index].x<.00001 && fcnC[index].y<.00001 && fcnC[index].x>-.00001 && fcnC[index].y>-.00001)) {
	
	//printf("F(%d,%d,%.2f)...
	  printf("F(%g,%g,%g)=%.5f + i*%.5f  %d\n", (float) i/Y0, (float) j/X0, (float)k/Zp,
	                     fcnC[index].x, fcnC[index].y, index);
        }
      }
      //printf("\n");
    }  
    for(int j=-Nx/2+1; j<0; j++) {
      for(int i=0; i<Ny/2+1; i++) {
        int index = (i) + (Ny/2+1)*(j+Nx) + Nx*(Ny/2+1)*k;
	
	if(!(fcnC[index].x<.00001 && fcnC[index].y<.00001 && fcnC[index].x>-.00001 && fcnC[index].y>-.00001)) {

	  printf("F(%g,%g,%g)=%.5f + i*%.5f  %d\n", (float) i/Y0, (float) j/X0, (float)k/Zp, 
	                   fcnC[index].x, fcnC[index].y, index);
        }
      }
        
      //printf("\n");
    }
  } 
  for(int k=-Nz/2+1; k<0; k++) { 
    for(int j=0; j<Nx/2+1; j++) { 
      for(int i=0; i<Ny/2+1; i++) {  
	int index = i + (Ny/2+1)*(j) + Nx*(Ny/2+1)*(k+Nz);
	
	if(!(fcnC[index].x<.00001 && fcnC[index].y<.00001 && fcnC[index].x>-.00001 && fcnC[index].y>-.00001)) {
	
	//printf("F(%d,%d,%.2f)...
	  printf("F(%g,%g,%g)=%.5f + i*%.5f  %d\n", (float) i/Y0, (float) j/X0, (float)k/Zp,
	                     fcnC[index].x, fcnC[index].y, index);
        }
      }
      //printf("\n");
    }  
    for(int j=-Nx/2+1; j<0; j++) {
      for(int i=0; i<Ny/2+1; i++) {
        int index = (i) + (Ny/2+1)*(j+Nx) + Nx*(Ny/2+1)*(k+Nz);
	
	if(!(fcnC[index].x<.00001 && fcnC[index].y<.00001 && fcnC[index].x>-.00001 && fcnC[index].y>-.00001)) {

	  printf("F(%g,%g,%g)=%.5f + i*%.5f  %d\n", (float) i/Y0, (float) j/X0, (float)k/Zp, 
	                   fcnC[index].x, fcnC[index].y, index);
        }
      }
        
      //printf("\n");
    }
  }  
  free(fcnC);
} 

void getfcnZCOMPLEX_Covering(hipfftComplex* fcn_d, int nLinks, int nChains, int* ky, int* kx, float* kz_d) 
{
  hipfftComplex *fcnC; 
  float *kz;
  fcnC = (hipfftComplex*) malloc(sizeof(hipfftComplex)*Nz*nLinks*nChains);
  kz = (float*) malloc(sizeof(float)*Nz*nLinks);
  hipMemcpy(fcnC, fcn_d, sizeof(hipfftComplex)*Nz*nLinks*nChains, hipMemcpyDeviceToHost);
  hipMemcpy(kz, kz_d, sizeof(float)*nLinks*Nz, hipMemcpyDeviceToHost);  //SEG FAULT
  for(int n=0; n<nChains; n++) { 
    for(int p=0; p<nLinks; p++) { 
      for(int i=0; i<Nz; i++) {  
	int index = i + p*Nz + n*Nz*nLinks;
	
	if( ky[p+nLinks*n] == 11 && (kx[p+nLinks*n] == 11 || kx[p+nLinks*n] == 106) ) {
	//if(!(fcnC[index].x<.0001 && fcnC[index].y<.0001 && fcnC[index].x>-.0001 && fcnC[index].y>-.0001)) {
	
	  //printf("F(%d,%d,%.2f)...
	  printf("F(%g,%g,%g)=%.5f + i*%.5f  %d,%d,%d\n", (float) ky[p+nLinks*n]/Y0, (float) kx[p+nLinks*n]/X0, kz[i+p*Nz],
	                     fcnC[index].x, fcnC[index].y, i,p,n);
        }
      }      
    }
  }
  free(fcnC); free(kz);
}     

void getfcn_Covering(hipfftComplex* fcn_d, int nLinks, int nChains, int* ky, int* kx)
{
  hipfftComplex *fcnC;
  fcnC = (hipfftComplex*) malloc(sizeof(hipfftComplex)*nLinks*nChains*Nz);
  hipMemcpy(fcnC, fcn_d, sizeof(hipfftComplex)*nLinks*nChains*Nz, hipMemcpyDeviceToHost);
  for(int n=0; n<nChains; n++) { 
    for(int p=0; p<nLinks; p++) { 
      for(int i=0; i<Nz; i++) {  
	int index = i + p*Nz + n*Nz*nLinks;
	
	if(!(fcnC[index].x<.0001 && fcnC[index].y<.0001 && fcnC[index].x>-.0001 && fcnC[index].y>-.0001)) {
		
	  printf("F(%g,%g,%g)=%.5f + i*%.5f  %d\n", (float) ky[p+nLinks*n]/Y0, (float) kx[p+nLinks*n]/X0, 2*M_PI*(float)(i-Nz/2)/Nz,
	                     fcnC[index].x, fcnC[index].y, index);
        }
      }
      //printf("\n");
    }
  }    
    
  free(fcnC);
} 


void getfcn(float* fcn_d) {
  hipfftReal *fcn;
  fcn = (float*) malloc(sizeof(float)*Ny*Nx*Nz);
  hipMemcpy(fcn, fcn_d, sizeof(float)*Nx*Ny*Nz, hipMemcpyDeviceToHost);
  
  for(int k=0; k<Nz; k++) {  
   for(int j=0; j<Nx; j++) {
    for(int i=0; i<Ny; i++) {
      int index = i + Ny*j + Nx*Ny*k;
      printf("f(%.2fPI,%.2fPI)=%.3e ", 2*(float)(i-Ny/2)/Ny, 2*(float)(j-Nx/2)/Nx, fcn[index]);     
      }
      printf("\n");
    } printf("\n");
   } 
  free(fcn); 
}      

void getfcnComplexPadded(hipfftComplex* fcn_d)
{
  hipfftComplex *fcnC;
  
  fcnC = (hipfftComplex*) malloc(sizeof(hipfftComplex)*(Ny)*Nx*Nz);
  for(int i=0; i<Nx*Ny*Nz; i++) {
    fcnC[i].x = 0;
    fcnC[i].y = 0;
  }
  hipMemcpy(fcnC, fcn_d, sizeof(hipfftComplex)*(Ny)*Nx*Nz, hipMemcpyDeviceToHost);
  //for(int k=0; k<(Nz); k++) { 
    /*for(int j=0; j<Nx/2+1; j++) { 
      for(int i=0; i<Ny; i++) {  
	int index = i + (Ny)*(j); // Nx*(Ny/2+1)*k;
	
	if(!(fcnC[index].x<.001 && fcnC[index].y<.001 && fcnC[index].x>-.001 && fcnC[index].y>-.001)) {
	
	//printf("F(%d,%d,%.2f)...
	  printf("F(%d,%d)=%.5f+i*%.5f  %d\n", i, j, //2*M_PI*(float)(k-Nz/2)/Nz,
	                     fcnC[index].x, fcnC[index].y, index);
        }
      }
      //printf("\n");
    }  
    for(int j=-Nx/2+1; j<0; j++) {
      for(int i=0; i<Ny; i++) {
        int index = (i) + (Ny)*(j+Nx);// + Nx*(Ny/2+1)*k;
	
	if(!(fcnC[index].x<.001 && fcnC[index].y<.001 && fcnC[index].x>-.001 && fcnC[index].y>-.001)) {

	  printf("F(%d,%d)=%.5f+i*%.5f  %d\n", i, j, //2*M_PI*(float)(k-Nz/2)/Nz, 
	                   fcnC[index].x, fcnC[index].y, index);
        }
      }
        
      //printf("\n");
    } */
  //}  
  
  for(int i=0; i<Nx*Ny*Nz; i++) {
    if(!(fcnC[i].x<.001 && fcnC[i].y<.001 && fcnC[i].x>-.001 && fcnC[i].y>-.001)) 
      printf("F(%d)=%.5f+i*%.5f\n", i, fcnC[i].x, fcnC[i].y);
  }  
  
  free(fcnC);
}     
     
                                             
   

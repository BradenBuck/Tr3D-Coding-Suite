#include "hip/hip_runtime.h"
#include "reductions.h"

Red::Red(int N) : N_(N)
{
  hipMalloc(&value, sizeof(float));   hipMemset(value, 0., sizeof(float));
  hipMalloc(&dum, N_*sizeof(float));  hipMemset(dum, 0., N_*sizeof(float));

  work_sum = NULL;  nwork_sum = 0;

  using namespace hipcub;

  // Set up work space for a summation
  CubDebugExit(DeviceReduce::Sum(work_sum, nwork_sum, dum, value, N_));

  CachingDeviceAllocator  g_allocator(true);
  g_allocator.DeviceAllocate(&work_sum, nwork_sum);
  hipMalloc(&work_sum, nwork_sum);
  
  work_max = NULL;  nwork_max = 0;

  // Set up work space for finding max(abs(float array))
  CubDebugExit(DeviceReduce::Reduce(work_max, nwork_max, dum, value, N_, max_op, zero));

  g_allocator.DeviceAllocate(&work_max, nwork_max);
  hipMalloc(&work_max, nwork_max);

  hipFree(dum);
}

Red::~Red() {
  hipFree(work_sum);  hipFree(work_max);  hipFree(value);
}

// val = max(abs(rmom))
void Red::MaxAbs(float* rmom, float* val)
{
  using namespace hipcub;
  CubDebugExit(DeviceReduce::Reduce(work_max, nwork_max, rmom, val, N_, max_op, zero));
}

// val = Sum
void Red::Sum(float* rmom, float* val, bool boo)
{ using namespace hipcub;
  CubDebugExit(DeviceReduce::Sum(work_sum, nwork_sum, rmom, val, N_));
}

// val += Sum
void Red::Sum(float* rmom, float* val)
{ using namespace hipcub;
  hipMemcpy(value, val, sizeof(float), hipMemcpyDeviceToDevice);
  CubDebugExit(DeviceReduce::Sum(work_sum, nwork_sum, rmom, val, N_));
  acc<<<1,1>>> (val, value);
}
